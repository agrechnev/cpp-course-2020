// Cuda example add2 by Oleksiy Grechnyev
// This one uses classical memory management


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <vector>

// Kernel: This runs on the GPU (device) !
__global__
void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    printf("thread = %d/%d, block = %d/%d, index = %d/%d \n", threadIdx.x, blockDim.x, blockIdx.x, gridDim.x, index, stride);
    for (int i = index; i< n ; i += stride)
        y[i] += x[i];
}

// This runs on the CPU (host)
int main(){
    int n = 1 << 20; // 1024**2
    int nF = n*sizeof(float);
    
    // Create data in the CPU memory (host)
    std::vector<float> x(n), y(n);
    
    // Initialize host
    for (int i=0; i<n; ++i) {
        x[i] = 2.0f;
        y[i] = 1.0f;
    }
    
    // Allocate GPU (device) memory
    float *dX, *dY;
    hipMalloc(&dX, nF);
    hipMalloc(&dY, nF);
    
    // Copy Device->Host
    hipMemcpy(dX, x.data(), nF, hipMemcpyHostToDevice);
    hipMemcpy(dY, y.data(), nF, hipMemcpyHostToDevice);
    
    // Automatic block size and number of blocks for max speed
//     int blockSize = 256;
//     int numBlocks = (n+ blockSize -1) / blockSize;

    // Smaller values for the demo
    int blockSize = 4;
    int numBlocks = 3;
    
    // Add
    add<<<numBlocks, blockSize>>>(n, dX, dY);
    
    // Not needed in this example
//     cudaDeviceSynchronize();
    
    // Copy Host->Device
    hipMemcpy(y.data(), dY, nF, hipMemcpyDeviceToHost);
    
    // Check the result: should be 0
    double maxE=0;
    for (int i = 0; i< n ; ++i)
        maxE = std::fmax(maxE, std::fabs(y[i] - 3.0f));
    std::cout << "maxE = " << maxE << std::endl;
    
    // Free memory
    hipFree(dX);
    hipFree(dY);
    
    return 0;
}
