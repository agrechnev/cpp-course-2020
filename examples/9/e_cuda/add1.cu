// Cuda example add1 by Oleksiy Grechnyev
// This one uses cudaMallocManaged() : unified CPU/GPU memory


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// Kernel: This runs on the GPU (device) !
__global__
void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    printf("thread = %d/%d, block = %d/%d, index = %d/%d \n", threadIdx.x, blockDim.x, blockIdx.x, gridDim.x, index, stride);
    for (int i = index; i< n ; i += stride)
        y[i] += x[i];
}

// This runs on the CPU (host)
int main(){
    int n = 1 << 20; // 1024**2
    
    // Alloc unified CPU/GPU memory
    float *x, *y;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    
    // Initialize data
    for (int i = 0; i< n ; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
   
   
    // Automatic block size and number of blocks for max speed
//     int blockSize = 256;
//     int numBlocks = (n+ blockSize -1) / blockSize;

    // Smaller values for the demo
    int blockSize = 4;
    int numBlocks = 3;
    
    // Add
    add<<<numBlocks, blockSize>>>(n, x, y);
    
    // Wait for GPU, needed here because of cudaMallocManaged()
    hipDeviceSynchronize();
    
    // Check the result: should be 0
    float maxE=0;
    for (int i = 0; i< n ; ++i)
        maxE = std::fmax(maxE, std::fabs(y[i] - 3.0f));
    std::cout << "maxE = " << maxE << std::endl;
    
    // Free memory
    hipFree(x);
    hipFree(y);
    
    return 0;
}
